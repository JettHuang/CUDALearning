#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t displayDeviceInfo();

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{

    // Show GPU Information
    displayDeviceInfo();



    hipError_t cudaStatus;

    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    hipError_t cudaStatus;

    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}


hipError_t displayDeviceInfo()
{
    hipError_t cudaStatus;

    // GPU Information
    int deviceCount = 0;
    cudaStatus = hipGetDeviceCount(&deviceCount);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipGetDeviceCount() returned %d\n -->%s\n", (int)cudaStatus, hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    if (deviceCount <= 0) {
        return cudaErrorDevicesUnavailable;
    }

    int dev = 0, driverVersion = 0, runtimeVersion = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Device %d:\"%s\"\n", dev, deviceProp.name);
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf("  CUDA Driver Version / Runtime Version         %d.%d  /  %d.%d\n",
        driverVersion / 1000, (driverVersion % 100) / 10,
        runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("  CUDA Capability Major/Minor version number:   %d.%d\n",
        deviceProp.major, deviceProp.minor);
    printf("  Total amount of global memory:                %.2f MBytes (%llu bytes)\n",
        (float)deviceProp.totalGlobalMem / (1024 * 1024), deviceProp.totalGlobalMem);
    printf("  GPU Clock rate:                               %.0f MHz (%0.2f GHz)\n",
        deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
    printf("  Memory Bus width:                             %d-bits\n",
        deviceProp.memoryBusWidth);
    if (deviceProp.l2CacheSize)
    {
        printf("  L2 Cache Size:                            	%d bytes\n",
            deviceProp.l2CacheSize);
    }
    printf("  Max Texture Dimension Size (x,y,z)            1D=(%d),2D=(%d,%d),3D=(%d,%d,%d)\n",
        deviceProp.maxTexture1D, deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1]
        , deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
    printf("  Max Layered Texture Size (dim) x layers       1D=(%d) x %d,2D=(%d,%d) x %d\n",
        deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1],
        deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1],
        deviceProp.maxTexture2DLayered[2]);
    printf("  Total amount of constant memory               %lu bytes\n",
        deviceProp.totalConstMem);
    printf("  Total amount of shared memory per block:      %lu bytes\n",
        deviceProp.sharedMemPerBlock);
    printf("  Total number of registers available per block:%d\n",
        deviceProp.regsPerBlock);
    printf("  Wrap size:                                    %d\n", deviceProp.warpSize);
    printf("  Maximun number of thread per multiprocesser:  %d\n",
        deviceProp.maxThreadsPerMultiProcessor);
    printf("  Maximun number of thread per block:           %d\n",
        deviceProp.maxThreadsPerBlock);
    printf("  Maximun size of each dimension of a block:    %d x %d x %d\n",
        deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("  Maximun size of each dimension of a grid:     %d x %d x %d\n",
        deviceProp.maxGridSize[0],
        deviceProp.maxGridSize[1],
        deviceProp.maxGridSize[2]);
    printf("  Maximu memory pitch                           %lu bytes\n", deviceProp.memPitch);

    return hipSuccess;
}
